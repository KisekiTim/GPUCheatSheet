
#include <hip/hip_runtime.h>
#include <iostream>

//Compile code:nvcc -o CudaDLL.dll --shared CUDABmarkDLL.cu

/*
This program runs the code when called as a DLL from python.
The purpose is to demonstrate that running GPU code from a DLL
has no performance penalty compared to a standalone *.exe file.

This program will create a 1024 x 1024 array, add 5 to all the
elements and measure the time it took to do so.

This program must be used with 1_CtypesDLL.py
*/
extern "C" {
    
    //The GPU function to add 5.
    __global__ void AddStuff(int *dev_N_Array)
    {
        int index = threadIdx.x + blockIdx.x*blockDim.x;
        dev_N_Array[index] +=5;

    }
    
    
    //The dll function
    __declspec(dllexport) int sum(int a, int b) 
    {
        
        //Creating a 1024x1024 array.
        int N = 1024*1024;
        int *dev_N_Array;

        //Allocating the memory size
        hipMalloc((void**) &dev_N_Array, N*sizeof(int));
        hipMemset(dev_N_Array,0,N*sizeof(int));

        //Creating the timing events
        float elapsedTime;    
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start,0);
        
        //Run the GPU kernal
        AddStuff<<<1024*1024/32, 32>>>(dev_N_Array);

        //Stopping the timing events
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        
        //Output the measurement time
        std::cout <<"Calculation time: " << elapsedTime ;

        return a+b;
    }
}