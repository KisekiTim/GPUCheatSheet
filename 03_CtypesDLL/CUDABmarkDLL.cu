
#include <hip/hip_runtime.h>
#include <iostream>

//Compile code nvcc -o [kernalname].dll --shared [kernalcode].cu

extern "C" {
    
    __global__ void AddStuff(int *dev_N_Array)
    {
        int index = threadIdx.x + blockIdx.x*blockDim.x;
        dev_N_Array[index] +=5;

    }
    
    
    
    __declspec(dllexport) int sum(int *indata, int *outdata, int a, int b) 
{
    int N = 1024*1024;
    int *dev_N_Array;

    hipMalloc((void**) &dev_N_Array, N*sizeof(int));
    hipMemset(dev_N_Array,0,N*sizeof(int));

    float elapsedTime;    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    AddStuff<<<1024*1024/32, 32>>>(dev_N_Array);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    std::cout << elapsedTime ;

    for (int n = 0; n < a; n++)
    {
    outdata[n] = indata[n]*2;
    }

    return a+b;
}
}