
#include <hip/hip_runtime.h>
#include <iostream>


    __global__ void AddStuff(int *dev_N_Array)
    {
        int index = threadIdx.x + blockIdx.x*blockDim.x;
        dev_N_Array[index] +=5;

    }
    
    
    
int main(void) 
{
    int N = 1024*1024;
    int *dev_N_Array;

    hipMalloc((void**) &dev_N_Array, N*sizeof(int));
    hipMemset(dev_N_Array,0,N*sizeof(int));

    float elapsedTime;    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    AddStuff<<<1024*1024/32, 32>>>(dev_N_Array);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    std::cout << elapsedTime ;

}
