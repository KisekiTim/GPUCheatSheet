
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

//Compile Command: nvcc [filename].cu -o [filename].exe
__device__ int AddOne(int A)
{
    //Functions in CUDA or GPU code start with __device__
    A += 1; 
    return A;
}

__global__ void SetIndex(int *A)
{
    //GPU function that sets adds array index value to index and then adds 1
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    A[index] += index;
    A[index] = AddOne(A[index]);

}
void Verify(int *A,int N)
{
    for (int n=0;n < N; n++)
    {
        std::cout<< A[n] << "\n";
    }
}

class GlobalParameters
{
    public:
    int Blocks;
    int Threads;
    int xwidth;
    int ywidth;
    int zwidth;
    
};

class ArrayInt: public GlobalParameters
{
    public:
    int N;
    int *Host;
    int *Device;
    //int xwidth;
    //int ywidth;
    //int zwidth;
    //const char *Description;


    //Functions

    void SetModelSpec(GlobalParameters ModelSpec)
    {
        N = ModelSpec.xwidth * ModelSpec.ywidth * ModelSpec.zwidth;
        Host = (int*) malloc(N*sizeof(int));
        hipMalloc((void**) &Device, N*sizeof(int));
        xwidth = ModelSpec.xwidth;
        ywidth = ModelSpec.ywidth;
        zwidth = ModelSpec.zwidth;
    }

    void CopyGPU2CPU ()
    {
        hipMemcpy(Host, Device, N*sizeof(int), hipMemcpyDeviceToHost);
    }

    void CopyCPU2GPU ()
    {
        hipMemcpy( Device, Host, N*sizeof(int), hipMemcpyHostToDevice);
    }

    void GPUClear()
    {
        hipFree(Device);
    }

    void GPUInitialize()
    {
        hipMalloc((void**) &Device, N*sizeof(int));
    }
    
    void GPUZero()
    {
        hipMemset(Device, 0, N*sizeof(int));
    }

    void CPUClear()
    {
        free(Host);
    }

    void CPUInitialize()
    {
        Host = (int*) malloc(N*sizeof(int));
    }

    void CPUZero()
    {
        memset(Host, 0, N*sizeof(int));
    }


    void DumpTxt(const char *Description)
    {
        int* temparray;
        temparray = (int*) malloc(xwidth*ywidth*zwidth*sizeof(int));
        hipMemcpy(temparray, Device, xwidth*ywidth*zwidth*sizeof(int), hipMemcpyDeviceToHost);
        
        std::ofstream parafile;
        parafile.open(Description) ;
        for (int z = 0; z < zwidth; z++)
        {
            parafile << z << "\n";
            for (int y = 0; y < ywidth; y++)
            {
                for (int x = 0; x < xwidth; x++)
                {
                    parafile << Host[x + y * xwidth + z *xwidth*ywidth] << ",";
                }
                parafile << "," <<  y << ",,";
                for (int x = 0; x < xwidth; x++)
                {
                    parafile << temparray[x + y * xwidth + z *xwidth*ywidth] << ",";
                }
                parafile << "," <<  y << ",,";
                for (int x = 0; x < xwidth; x++)
                {
                    parafile << Host[x + y * xwidth + z *xwidth*ywidth]-temparray[x + y * xwidth + z *xwidth*ywidth] << ",";
                }
                parafile << "," <<  y << ",,";
                
                for (int x = 0; x < xwidth; x++)
                {
                    parafile << x + y * xwidth + z *xwidth*ywidth  << ",";
                }
                parafile << "," <<  y << ",,";
                parafile << "\n";

            }
            parafile << z << "\n";
            for (int x = 0; x < xwidth; x++)
                {
                    parafile << x << ",";
                }
            parafile << ","  << ",,";
            for (int x = 0; x < xwidth; x++)
                {
                    parafile << x << ",";
                }
            parafile << ","  << ",,";
            for (int x = 0; x < xwidth; x++)
                {
                    parafile << x << ",";
                }
            parafile << ","  << ",,";
            for (int x = 0; x < xwidth; x++)
                {
                    parafile << x << ",";
                }

                parafile << "\n";
                
        }
        free(temparray);
    }

};

class GPUFunction: public GlobalParameters
{
    public:
    int N;
    ArrayInt AddArray;
    

    void SetIndexP1(GlobalParameters ModelSpec)
    {
        N = AddArray.N;
        AddArray.CopyCPU2GPU();
        SetIndex<<<ModelSpec.Blocks,ModelSpec.Threads>>>(AddArray.Device);
        hipDeviceSynchronize();
        AddArray.CopyGPU2CPU();
    }

};


int main(void)
{
    //Set global model size and GPU kernel parameters
    GlobalParameters ModelSpec;
    ModelSpec.Blocks =5;
    ModelSpec.Threads= 4;
    ModelSpec.xwidth =20;
    ModelSpec.ywidth =1;
    ModelSpec.zwidth = 1;

    //Create CPU/GPU Array object
    ArrayInt A;
    A.SetModelSpec(ModelSpec);

    //Set some initial values to the CPU array
    for (int n = 0; n < 20 ; n++)
    {
        A.Host[n] = 1;
    }

    //Create GPU function object
    GPUFunction AddGPU;

    //Set GPU function input Array
    AddGPU.AddArray = A;

    //Run the GPU method
    AddGPU.SetIndexP1(ModelSpec);

    //Copy back the GPU attribute
    A = AddGPU.AddArray;
    
    //Verify
    Verify(A.Host,20);

}